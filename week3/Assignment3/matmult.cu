#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#define BLOCK_SIZE 32
//Sequential version: One thread does it all. AKA--> Launch configuration <<<1,1,>>>
__global__
void gpu1(int m, int n, int k, double* d_A, double* d_B, double* d_C ){
    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
            d_C[i*(n) + j] = 0.0;
            for (int rc = 0; rc < k; rc++){
                d_C[i*(n) + j] += d_A[i*(k) + rc] * d_B[rc*(n) + j];
            }
        }
    }
}

//Naive version: One thread per element in C
__global__
void gpu2( int m, int n, int k, double*  d_A, double* d_B, double* d_C) {

    int i = blockIdx.y*blockDim.y+threadIdx.y;  //column thead id
    int j = blockIdx.x*blockDim.x+threadIdx.x;  //row thread id
    
    if (j<m && i<n) {
        d_C[i*n + j] = 0.0;
        for (int rc = 0; rc < k; rc++){
            d_C[i*n + j] += d_A[i*k + rc] * d_B[rc*n + j];
        }
    }
}

//one thread every 2 elements of C
__global__
void gpu3(int m, int n, int k, double* d_A, double* d_B, double* d_C ){
    int i = blockIdx.y*blockDim.y+threadIdx.y;  //column thead id
    int j = blockIdx.x*2*blockDim.x+threadIdx.x;  //row thread id
    int j2 = j + blockDim.x;

    if (j<m && i<n && j2 < m) {
        d_C[i*n + j2] = 0.0;
        d_C[i*n + j] = 0.0;
        for (int rc = 0; rc < k; rc++){
            d_C[i*n + j2] += d_A[i*k + rc] * d_B[rc*n + j2];
            d_C[i*n + j] += d_A[i*k + rc] * d_B[rc*n + j];
        }
    } else if (j<m && i<n) {
        d_C[i*n + j] = 0.0;
        for (int rc = 0; rc < k; rc++){
            d_C[i*n + j] += d_A[i*k + rc] * d_B[rc*n + j];
        }
    }
}

//one thread every 4 elements of C
__global__
void gpu4(int m, int n, int k, double* d_A, double* d_B, double* d_C ){
    int i = blockIdx.y*blockDim.y+threadIdx.y;  //column thead id
    int j = blockIdx.x*4*blockDim.x+threadIdx.x;  //row thread id
    int j2 = j + blockDim.x;
    int j3 = j2 + blockDim.x;
    int j4 = j3 + blockDim.x;

    if (j<m && i<n && j2 < m && j3 < m && j4 < m) {
        d_C[i*n + j4] = 0.0;
        d_C[i*n + j3] = 0.0;
        d_C[i*n + j2] = 0.0;
        d_C[i*n + j] = 0.0;
        for (int rc = 0; rc < k; rc++){
            d_C[i*n + j4] += d_A[i*k + rc] * d_B[rc*n + j4];
            d_C[i*n + j3] += d_A[i*k + rc] * d_B[rc*n + j3];
            d_C[i*n + j2] += d_A[i*k + rc] * d_B[rc*n + j2];
            d_C[i*n + j] += d_A[i*k + rc] * d_B[rc*n + j];
        }
    } else if (j<m && i<n && j3 < m) {
        d_C[i*n + j3] = 0.0;
        d_C[i*n + j2] = 0.0;
        d_C[i*n + j] = 0.0;
        for (int rc = 0; rc < k; rc++){
            d_C[i*n + j3] += d_A[i*k + rc] * d_B[rc*n + j3];
            d_C[i*n + j2] += d_A[i*k + rc] * d_B[rc*n + j2];
            d_C[i*n + j] += d_A[i*k + rc] * d_B[rc*n + j];
        }
    } else if (j<m && i<n && j2 < m) {
        d_C[i*n + j2] = 0.0;
        d_C[i*n + j] = 0.0;
        for (int rc = 0; rc < k; rc++){
            d_C[i*n + j2] += d_A[i*k + rc] * d_B[rc*n + j2];
            d_C[i*n + j] += d_A[i*k + rc] * d_B[rc*n + j];
        }
    } else if (j<m && i<n) {
        d_C[i*n + j] = 0.0;
        for (int rc = 0; rc < k; rc++){
            d_C[i*n + j] += d_A[i*k + rc] * d_B[rc*n + j];
        }
    }
}

//blocked version with shared memory
__global__
void gpu5(int m, int n, int k, double* d_A, double* d_B, double* d_C ){
    __shared__ double A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double B[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double C[BLOCK_SIZE][BLOCK_SIZE];
    int i = blockIdx.y*blockDim.y+threadIdx.y;  //column thead id
    int j = blockIdx.x*blockDim.x+threadIdx.x;  //row thread id

    C[threadIdx.x][threadIdx.y] = 0.0;
    for (int rc = 0; rc < k; rc+=BLOCK_SIZE){

        A[threadIdx.y][threadIdx.x] = d_A[i*k + rc + threadIdx.x];
        B[threadIdx.y][threadIdx.x] = d_B[(rc + threadIdx.y)*n + j];
        
        __syncthreads();

        for (int rc2 = 0; rc2 < BLOCK_SIZE; rc2++){
            C[threadIdx.y][threadIdx.x] += A[threadIdx.y][rc2] * B[rc2][threadIdx.x];
        }
        __syncthreads();
    }
    d_C[i*n + j] = C[threadIdx.y][threadIdx.x];
}

extern "C" {
    #include <cblas.h>
    void matmult_lib(int m, int n, int k,double* A, double* B, double* C){
        cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, 1.0, A, k, B, n, 0.0, C, n);
        //cblas_dgemm(CblasColMajor, CblasNoTrans, CblasNoTrans, n, m, k, 1.0, B, n, A, k, 0.0, C, n);
    }

    void matmult_gpulib(int m, int n, int k,double* A, double* B, double* C){
        double* d_A; hipMalloc((void**)&d_A, m*k*sizeof(double));
        double* d_B; hipMalloc((void**)&d_B, k*n*sizeof(double));
        double* d_C; hipMalloc((void**)&d_C, m*n*sizeof(double));

        // Transfer data from host to device
        hipMemcpy(d_A, A, m*k*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, k*n*sizeof(double), hipMemcpyHostToDevice);

        // Do the actual multiplication
        hipblasDgemm('n', 'n', n, m, k, 1.0, d_B, n, d_A, k, 0.0, d_C, n);
        checkCudaErrors(hipDeviceSynchronize());

        // Transfer results from device to host
        hipMemcpy(C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

        // Free memory
        hipFree(d_C);
        hipFree(d_A);
        hipFree(d_B);
    }

    void matmult_gpu5(int m, int n, int k, double* h_A, double* h_B, double* h_C){
        double* d_A; hipMalloc((void**)&d_A, m*k*sizeof(double));
        double* d_B; hipMalloc((void**)&d_B, k*n*sizeof(double));
        double* d_C; hipMalloc((void**)&d_C, m*n*sizeof(double));

        // Transfer data from host to device
        hipMemcpy(d_A, h_A, m*k*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, k*n*sizeof(double), hipMemcpyHostToDevice);

        dim3 NUM_BLOCKS = dim3(m/32, n/32, 1);
        dim3 NUM_THREADS = dim3(32, 32, 1);

        // Kernel launch
        gpu5<<<NUM_BLOCKS, NUM_THREADS>>>(m, n, k, d_A, d_B, d_C);
        checkCudaErrors(hipDeviceSynchronize());

        // Transfer results from device to host
        hipMemcpy(h_C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

        // Free memory
        hipFree(d_C);
        hipFree(d_A);
        hipFree(d_B);
    }

    void matmult_gpu4(int m, int n, int k, double* h_A, double* h_B, double* h_C){
        double* d_A; hipMalloc((void**)&d_A, m*k*sizeof(double));
        double* d_B; hipMalloc((void**)&d_B, k*n*sizeof(double));
        double* d_C; hipMalloc((void**)&d_C, m*n*sizeof(double));

        // Transfer data from host to device
        hipMemcpy(d_A, h_A, m*k*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, k*n*sizeof(double), hipMemcpyHostToDevice);

        int blockx = 0, blocky = 0;

        if(m%128)
            blockx = 1;
        if (n%32)
            blocky = 1; 
        dim3 NUM_BLOCKS = dim3(m/128 + blockx, n/32 + blocky, 1);
        dim3 NUM_THREADS = dim3(32, 32, 1);

        if (m*n/4 <= 1024){
            NUM_BLOCKS = dim3(1, 1, 1);
            blockx = 0;
            if (m%4)
                blockx = 1;
            NUM_THREADS = dim3(m/4 + blockx, n, 1);
        }

        // Kernel launch
        gpu4<<<NUM_BLOCKS, NUM_THREADS>>>(m, n, k, d_A, d_B, d_C);
        checkCudaErrors(hipDeviceSynchronize());

        // Transfer results from device to host
        hipMemcpy(h_C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

        // Free memory
        hipFree(d_C);
        hipFree(d_A);
        hipFree(d_B);
    }

    void matmult_gpu3(int m, int n, int k, double* h_A, double* h_B, double* h_C){
        double* d_A; hipMalloc((void**)&d_A, m*k*sizeof(double));
        double* d_B; hipMalloc((void**)&d_B, k*n*sizeof(double));
        double* d_C; hipMalloc((void**)&d_C, m*n*sizeof(double));

        // Transfer data from host to device
        hipMemcpy(d_A, h_A, m*k*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, k*n*sizeof(double), hipMemcpyHostToDevice);

        int blockx = 0, blocky = 0;
        if(m%64)
            blockx = 1;
        if (n%32)
            blocky = 1;
        dim3 NUM_BLOCKS = dim3(m/64 + blockx, n/32 + blocky, 1);
        dim3 NUM_THREADS = dim3(32, 32, 1);

        if (m*n/2 <= 1024){
            NUM_BLOCKS = dim3(1, 1, 1);
            NUM_THREADS = dim3(m/2 + m % 2, n, 1);
        }

        // Kernel launch
        gpu3<<<NUM_BLOCKS, NUM_THREADS>>>(m, n, k, d_A, d_B, d_C);
        checkCudaErrors(hipDeviceSynchronize());

        // Transfer results from device to host
        hipMemcpy(h_C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

        // Free memory
        hipFree(d_C);
        hipFree(d_A);
        hipFree(d_B);
    }

    void matmult_gpu2(int m, int n, int k, double* h_A, double* h_B, double* h_C){
        double* d_A; hipMalloc((void**)&d_A, m*k*sizeof(double));
        double* d_B; hipMalloc((void**)&d_B, k*n*sizeof(double));
        double* d_C; hipMalloc((void**)&d_C, m*n*sizeof(double));

        // Transfer data from host to device
        hipMemcpy(d_A, h_A, m*k*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, k*n*sizeof(double), hipMemcpyHostToDevice);

        int blockx = 0, blocky = 0;
        if(m%32)
            blockx = 1;
        if (n%32)
            blocky = 1;
        dim3 NUM_BLOCKS = dim3(m/32 + blockx, n/32 + blocky, 1);
        dim3 NUM_THREADS = dim3(32, 32, 1);

        if (m*n <= 1024){
            NUM_BLOCKS = dim3(1, 1, 1);
            NUM_THREADS = dim3(m, n, 1);
        }

        // Kernel launch
        gpu2<<<NUM_BLOCKS, NUM_THREADS>>>(m, n, k, d_A, d_B, d_C);
        checkCudaErrors(hipDeviceSynchronize());

        // Transfer results from device to host
        hipMemcpy(h_C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

        // Free memory
        hipFree(d_C);
        hipFree(d_A);
        hipFree(d_B);
    }

    void matmult_gpu1(int m, int n, int k, double* h_A, double* h_B, double* h_C ){
        double* d_A; hipMalloc((void**)&d_A, m*k*sizeof(double));
        double* d_B; hipMalloc((void**)&d_B, k*n*sizeof(double));
        double* d_C; hipMalloc((void**)&d_C, m*n*sizeof(double));
        
        // Transfer data from host to device
        hipMemcpy(d_A, h_A, m*k*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, k*n*sizeof(double), hipMemcpyHostToDevice);

        // Kernel launch
        gpu1<<<1,1>>>(m, n, k, d_A, d_B, d_C);
        checkCudaErrors(hipDeviceSynchronize());

        // Transfer results from device to host
        hipMemcpy(h_C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

        // Free memory
        hipFree(d_C);
        hipFree(d_A);
        hipFree(d_B);
    }
} // end extern C
