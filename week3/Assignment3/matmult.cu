#include "hip/hip_runtime.h"
//Sequential version: One thread does it all. AKA--> Launch configuration <<<1,1,>>>
__global__
void gpu1(int *m, int *n, int *k, double* d_A, double* d_B, double* d_C ){
    for (int i = 0; i < *m; i++){
        for (int j = 0; j < *n; j++){
            d_C[i*(*n) + j] = 0;
            for (int rc = 0; rc < *k; rc++){
                d_C[i*(*n) + j] += d_A[i*(*k) + rc] * d_B[rc*(*n) + j];
            }
        }
    }
}

void matmult_gpu1(int m, int n, int k, double* h_A, double* h_B, double* h_C ){

    double* d_A; hipMalloc((void**)&d_A, m*k*sizeof(double));
    double* d_B; hipMalloc((void**)&d_B, k*n*sizeof(double));
    double* d_C; hipMalloc((void**)&d_C, m*n*sizeof(double));
    int *d_m, *d_n, *d_k;

    // Transfer data from host to device
    hipMemcpy(d_m, &m, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_k, &k, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_A, h_A, m*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k*n*sizeof(double), hipMemcpyHostToDevice);

    // Kernel launch
    gpu1<<<1,1>>>(d_m, d_n, d_k, d_A, d_B, d_C);
    checkCudaErrors(hipDeviceSynchronize());

    // Transfer results from device to host
    hipMemcpy(h_C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

    //print matrix d_C
    //print_matrix( m,  n, d_C);

    // Free memory
    hipFree(d_C);
    hipFree(d_A);
    hipFree(d_B);

}