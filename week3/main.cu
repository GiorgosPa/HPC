#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

__global__
void hello(){
    int block, blocks, thread, gthread, threads_per_block;
    block = blockIdx.x;
    thread = threadIdx.x;
    threads_per_block = blockDim.x;
    gthread = block * threads_per_block + thread;
    blocks = gridDim.x;
    if (gthread == 100)
    {
        // segmentation fault
        int *a = (int*) 0x10000;
        *a = 0;
    }
    printf("Hello world! I’m thread %d out of %d in block %d. My global thread id is %d out of %d\n",
     thread, threads_per_block, block, gthread, threads_per_block*blocks);
}


int main(){
    hello<<<16,16>>>();
    checkCudaErrors(hipDeviceSynchronize());
    return(0);
}