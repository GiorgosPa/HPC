
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void hello(){
    int block, thread, gthread, threads_per_block;
    block = blockIdx.x;
    thread = threadIdx.x;
    threads_per_block = blockDim.x;
    gthread = block * threads_per_block + thread;
    printf("Hello world! I’m thread %d out of %d in block %d. My global thread id is %d out of %d\n",
     thread, threads_per_block, block, gthread, threads_per_block);
}


int main(){
    return(0);
}