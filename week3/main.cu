#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void hello(){
    int block, blocks, thread, gthread, threads_per_block;
    block = blockIdx.x;
    thread = threadIdx.x;
    threads_per_block = blockDim.x;
    gthread = block * threads_per_block + thread;
    blocks = gridDim.x;
    printf("Hello world! I’m thread %d out of %d in block %d. My global thread id is %d out of %d\n",
     thread, threads_per_block, block, gthread, threads_per_block*);
}


int main(){
    hello<<<16,16>>>();
    hipDeviceSynchronize();
    return(0);
}