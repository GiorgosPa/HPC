#include <stdio.h>
#include <stdlib.h>
#include "writepng.h"
#include "mandel.h"
#include <hip/hip_runtime_api.h>


int
main(int argc, char *argv[]) {

    int   width, height;
    int   *d_width, *d_height;
    int	  max_iter;
    int   *d_max_iter;
    int   *image;
    int   *d_image;

    width    = 2601;
    height   = 2601;
    max_iter = 400;

    hipMalloc((void**) &d_width, sizeof(int));
    hipMalloc((void**) &d_height, sizeof(int));
    hipMalloc((void**) &d_max_iter, sizeof(int));
    
    hipMemcpy(d_width, &width, width * height * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(d_height, &height, width * height * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(d_max_iter, &max_iter, width * height * sizeof(int), hipMemcpyDeviceToHost);

    // command line argument sets the dimensions of the image
    if ( argc == 2 ) width = height = atoi(argv[1]);

    image = (int *)malloc( width * height * sizeof(int));

    if ( image == NULL ) {
       fprintf(stderr, "memory allocation failed!\n");
       return(1);
    }
    hipMalloc((void**) &d_image, width * height * sizeof(int));

    dim3 blocks = dim3(64,64);
    dim3 threads = dim3(64,64);

    mandel<<<17, 153>>>(d_width, d_height, d_image, d_max_iter);

    checkCudaErrors(hipDeviceSynchronize());
    hipMemcpy(image, d_image, width * height * sizeof(int), hipMemcpyDeviceToHost);
    writepng("mandelbrot.png", image, width, height);

    return(0);
}
