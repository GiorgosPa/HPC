#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "writepng.h"
#include "mandel.h"
#include <hip/hip_runtime_api.h>
#include <math.h>

int
main(int argc, char *argv[]) {

    int   width, height;
    int   *d_width, *d_height;
    int	  max_iter;
    int   *d_max_iter;
    int   *image;
    int   *d_image;
    int   k = 32;

    width    = 2601;
    height   = 2601;
    max_iter = 400;

    // command line argument sets the dimensions of the image
    if ( argc == 2 ) width = height = atoi(argv[1]);

    hipSetDevice(6);

    hipMalloc((void**) &d_width, sizeof(int));
    hipMalloc((void**) &d_height, sizeof(int));
    hipMalloc((void**) &d_max_iter, sizeof(int));
    
    hipMemcpy(d_width, &width, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_height, &height, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_max_iter, &max_iter, sizeof(int), hipMemcpyHostToDevice);

    image = (int *)malloc( width * height * sizeof(int));

    if ( image == NULL ) {
       fprintf(stderr, "memory allocation failed!\n");
       return(1);
    }
    hipMalloc((void**) &d_image, width * height * sizeof(int));

    int blockx = ceil(width / (double)k);
    int blocky = ceil(height / (double)k); 
    
    dim3 blocks = dim3(blockx,blocky,1);
    dim3 threads = dim3(k,k,1);

    mandel<<<blocks, threads>>>(d_width, d_height, d_image, d_max_iter);

    checkCudaErrors(hipDeviceSynchronize());
    hipMemcpy(image, d_image, width * height * sizeof(int), hipMemcpyDeviceToHost);
    writepng("mandelbrot.png", image, width, height);

    return(0);
}
