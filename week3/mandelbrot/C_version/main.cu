#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "writepng.h"


__global__
void mandel(int disp_width, int disp_height, int *array, int max_iter) {

    double  scale_real, scale_imag;
    double  x, y, u, v, u2, v2;
    int     iter;

    scale_real = 3.5 / (double)disp_width;
    scale_imag = 3.5 / (double)disp_height;

    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;
    if ((tidx >= disp_width) || (tidy >= disp_height))
        return;

    //for(i = 0; i < disp_width; i++) {

        x = ((double)tidx * scale_real) - 2.25;

//      for(j = 0; j < disp_height; j++) {
            y = ((double)tidy * scale_imag) - 1.75;

            u    = 0.0;
            v    = 0.0;
            u2   = 0.0;
            v2   = 0.0;
            iter = 0;

            while ( u2 + v2 < 4.0 &&  iter < max_iter ) {
            v = 2 * v * u + y;
            u = u2 - v2 + x;
            u2 = u*u;
            v2 = v*v;
            iter = iter + 1;
            }

            // if we exceed max_iter, reset to zero
            iter = iter == max_iter ? 0 : iter;

            //array[i*disp_height + j] = iter;
            array[tidx*disp_height + tidy] = iter;

//      }
//    }
}



int
main(int argc, char *argv[]) {

    int   width, height;
    int	  max_iter;
    int   *image;
    int   *d_image;

    width    = 2601;
    height   = 2601;
    max_iter = 400;

    // command line argument sets the dimensions of the image
    if ( argc == 2 ) width = height = atoi(argv[1]);

    image = (int *)malloc( width * height * sizeof(int));

    if ( image == NULL ) {
       fprintf(stderr, "memory allocation failed!\n");
       return(1);
    }
    hipMalloc((void**) &d_image, width * height * sizeof(int));
    dim3 blocks = dim3(64,64);
    dim3 threads = dim3(64,64);

    mandel<<<blocks, threads>>>(width, height, d_image, max_iter);

    hipMemcpyDeviceToHost(image, d_image, width * height * sizeof(int), hipMemcpyDeviceToHost);
    writepng("mandelbrot.png", image, width, height);

    return(0);
}
