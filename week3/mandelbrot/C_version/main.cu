#include <stdio.h>
#include <stdlib.h>
#include "writepng.h"
#include "mandel.h"


int
main(int argc, char *argv[]) {

    int   width, height;
    int	  max_iter;
    int   *image;
    int   *d_image;

    width    = 2601;
    height   = 2601;
    max_iter = 400;

    // command line argument sets the dimensions of the image
    if ( argc == 2 ) width = height = atoi(argv[1]);

    image = (int *)malloc( width * height * sizeof(int));

    if ( image == NULL ) {
       fprintf(stderr, "memory allocation failed!\n");
       return(1);
    }
    hipMalloc((void**) &d_image, width * height * sizeof(int));
    dim3 blocks = dim3(64,64);
    dim3 threads = dim3(64,64);

    mandel<<<blocks, threads>>>(width, height, d_image, max_iter);

    hipMemcpy(image, d_image, width * height * sizeof(int), hipMemcpyDeviceToHost);
    writepng("mandelbrot.png", image, width, height);

    return(0);
}
