
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__
void mandel(int *disp_width, int *disp_height, int *array, int *max_iter) {

    double  scale_real, scale_imag;
    double  x, y, u, v, u2, v2;
    int     i,j,iter;

    scale_real = 3.5 / (double)*disp_width;
    scale_imag = 3.5 / (double)*disp_height;

    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("thread %d\n", tidx);
    //int tidy = blockIdx.y * blockDim.y + threadIdx.y;
    //if ((tidx >= disp_width) || (tidy >= disp_height))
    //   return;

    //for(i = 0; i < disp_width; i++) {

        x = ((double)tidx * scale_real) - 2.25;
        //x = ((double)i * scale_real) - 2.25;
        
      for(j = 0; j < *disp_height; j++) {
            //y = ((double)tidy * scale_imag) - 1.75;
            y = ((double)j * scale_imag) - 1.75;

            u    = 0.0;
            v    = 0.0;
            u2   = 0.0;
            v2   = 0.0;
            iter = 0;

            while ( u2 + v2 < 4.0 &&  iter < *max_iter ) {
                v = 2 * v * u + y;
                u = u2 - v2 + x;
                u2 = u*u;
                v2 = v*v;
                iter = iter + 1;
            }

            // if we exceed max_iter, reset to zero
            iter = iter == *max_iter ? 0 : iter;
            array[i*(*disp_height)+ j] = iter;
            //array[tidx*disp_height + tidy] = iter;

      }
    //}
}