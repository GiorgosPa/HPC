#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

//Sequential version: One thread does it all. AKA--> Launch configuration <<<1,1,>>>
__global__
void gpu1(int m, int n, int k, double* d_A, double* d_B, double* d_C ){
    double temp;
    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
                 temp = 0;
            for (int rc = 0; rc < k; rc++){
                //C[i][j] += A[i][rc] * B[rc][j];
                //temp += d_A[i*k + rc] * d_B[rc*n + j]; //original
                temp += d_A[i*k + rc] * d_B[rc*n + j];

            }
            //d_C[i*n + j] = temp; //original
            d_C[i*n + j] = temp;
        }
    }
}


extern "C" {
    #include <cblas.h>
void matmult_gpu1(int m, int n, int k, double* h_A, double* h_B, double* h_C ){
          hipSetDevice(3);
          double* d_A; hipMalloc((void**)&d_A, m*k*sizeof(double));
          double* d_B; hipMalloc((void**)&d_B, k*n*sizeof(double));
          double* d_C; hipMalloc((void**)&d_C, m*n*sizeof(double));
          

// Transfer data from host to device
hipMemcpy(d_A, h_A, m*k*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B, k*n*sizeof(double), hipMemcpyHostToDevice);

// Kernel launch
gpu1<<<1,1>>>(m, n, k, d_A, d_B, d_C);
checkCudaErrors(hipDeviceSynchronize());

 // Transfer results from device to host
 hipMemcpy(h_C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

//print matrix d_C
//print_matrix( m,  n, d_C);

 // Free memory
 hipFree(d_C);
 hipFree(d_A);
 hipFree(d_B);

}


void print_matrix(int m, int n, double* A){
	for(int i=0; i<m; i++){
	  for(int j=0; j<n; j++){
	      printf("%f\t", A[i + j]);
	  }
	  printf("\n");
	}
}






/*
//Naive version: One thread per element in C
__global__
void matmult_gpu2( double*  d_A, double* d_B, double* d_C, int DIM) {
  //where DIM is the size of the N by N (A, B, and C) square matrices


    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    double d_tempSum = 0;  //temporarily holds row*column sum of its respective block

    // each thread computes one element of the block (sub-matrix)
    for (int i = 0; i < DIM; i++) {
        d_tempSum += d_A[row * DIM + i] * d_B[i * DIM + col];
    }

    d_C[row * DIM + col] = d_tempSum;
}


//Improved Naive version: Each thread computes exactly 2 elements of C
__global__
void matmult_gpu3(double* d_A, double* d_B, double* d_C, ){

}

//Improved Naive version 2: Each thread computes exactly 4 elements of C
__global__
void matmult_gpu4(double* d_A, double* d_B, double* d_C, ){

}

//Shared Memory version: loads A and B (or segments of them) into shared memory to improve performance
__global__
void matmult_gpu5(double* d_A, double* d_B, double* d_C, ){

}

//non-CUDA version
    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
        	C[i][j] = 0.0;
        }
    }

    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
            for (int rc = 0; rc < k; rc++){
                C[i][j] += A[i][rc] * B[rc][j];
            }
        }
    }
}

*/
}
