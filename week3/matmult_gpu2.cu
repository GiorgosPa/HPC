#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>




//Naive version: One thread per element in C
__global__
void gpu2( int m, int n, int k, double*  d_A, double* d_B, double* d_C) {

    int i = blockIdx.y*blockDim.y+threadIdx.y;  //row thead id
    int j = blockIdx.x*blockDim.x+threadIdx.x;  //column thread id

    if (i<m && j<n) {
        for (int rc = 0; rc < k; rc++){
            d_C[i*n + j] += d_A[i*k + rc] * d_B[rc*n + j];
            //atomicADD( &d_C[i*n + j] , d_A[i*k + rc] * d_B[rc*n + j] );
        }
    }

}

extern "C" {
    #include <cblas.h>
void matmult_gpu2(int m, int n, int k, double* h_A, double* h_B, double* h_C){

          double* d_A; hipMalloc((void**)&d_A, m*k*sizeof(double));
          double* d_B; hipMalloc((void**)&d_B, k*n*sizeof(double));
          double* d_C; hipMalloc((void**)&d_C, m*n*sizeof(double));
          int m;
          int n;
          int k;

// Transfer data from host to device
hipMemcpy(d_C, h_C, m*n*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(d_A, h_A, m*k*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B, k*n*sizeof(double), hipMemcpyHostToDevice);

if (m*n <= 1024) {
    NUM_BLOCKS = dim3(1, 1, 1);
    NUM_THREADS =  dim3(32, 32, 1);
}

//Q: IS THIS OKAY? one thread per element in d_C(mxn)
NUM_BLOCKS = dim3(m/32, n/32, 1);
NUM_THREADS =  dim3(32, 32, 1);

 // Kernel launch
 gpu2<<<NUM_BLOCKS, NUM_THREADS>>>(m, n, k, d_A, d_B, d_C);
 checkCudaErrors(hipDeviceSynchronize());

 // Transfer results from device to host
 hipMemcpy(h_C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

//print matrix d_C
//print_matrix( m,  n, d_C);

 // Free memory
 hipFree(d_C);
 hipFree(d_A);
 hipFree(d_B);

}

} /* from extern "C" */
