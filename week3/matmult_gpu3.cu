#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>




//Naive version: Each thread computes exactly 2 threads of matrix C
__global__
void gpu3( int m, int n, int k, double*  d_A, double* d_B, double* d_C) {

    int j = blockIdx.y*blockDim.y+threadIdx.y;  //col thread id 
    int i = blockIdx.x*2*blockDim.x+threadIdx.x;  //row thread id
    int i2 = i + blockDim.x;  //row thread id

    double temp = 0;
    double temp2 = 0;

    if (i<m && i2<m && j<n) {
        for (int rc = 0; rc < k; rc++){
            temp += d_A[i*k + rc] * d_B[rc*n + j];
            temp2 += d_A[i2*k + rc] * d_B[rc*n + j];

        }
        d_C[i*n + j]  = temp;
        d_C[i2*n + j]  = temp2;
    }

    temp = 0;
    if (i<m && j<n) {
        for (int rc = 0; rc < k; rc++){
            temp += d_A[i*k + rc] * d_B[rc*n + j];
        }
        d_C[i*n + j]  = temp;
    } 


}

extern "C" {
    #include <cblas.h>
void matmult_gpu3(int m, int n, int k, double* h_A, double* h_B, double* h_C){
          hipSetDevice(3);
          double* d_A; hipMalloc((void**)&d_A, m*k*sizeof(double));
          double* d_B; hipMalloc((void**)&d_B, k*n*sizeof(double));
          double* d_C; hipMalloc((void**)&d_C, m*n*sizeof(double));


// Transfer data from host to device
hipMemcpy(d_A, h_A, m*k*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B, k*n*sizeof(double), hipMemcpyHostToDevice);

//Q: IS THIS OKAY? one thread per element in d_C(mxn)
dim3 NUM_BLOCKS = dim3(m/64 + 1, n/32 + 1, 1);
dim3 NUM_THREADS =  dim3(32, 32, 1);

if (m*n/2 <= 1024) {
    NUM_BLOCKS = dim3(1, 1, 1);
    NUM_THREADS =  dim3(m/2 + 1, n, 1);
}


 // Kernel launch
 gpu3<<<NUM_BLOCKS, NUM_THREADS>>>(m, n, k, d_A, d_B, d_C);
 checkCudaErrors(hipDeviceSynchronize());

 // Transfer results from device to host
 hipMemcpy(h_C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

//print matrix d_C
//print_matrix( m,  n, d_C);

 // Free memory
 hipFree(d_C);
 hipFree(d_A);
 hipFree(d_B);

}

} /* from extern "C" */
