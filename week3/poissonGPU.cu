#include "hip/hip_runtime.h"
#include "poissonGPU.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

//when we have memory latency issues, it's because we have not enough threads (from nvcc profiler analysis)
//In Kernel Memory (guided): it would light up red if there's a bottleneck in the bandwidth (eg. L1 L2 cache etc.)

/* Jaccobi GPU Method : one thread does all <<1,1>> */
__global__
void jaccobi(int N, double delta, double* d_f, double* d_u, double* d_uold){
	for(int i=1; i<N-1; i++){
		for(int j=1; j<N-1; j++){
			d_u[i*N +j] = 0.25 *(d_uold[(i-1)*N +j] + d_uold[(i+1)*N +j] + d_uold[i*N + j-1] + d_uold[i*N + j+1] + delta*delta*d_f[i*N +j]);
		}
	}
}


__global__
void jaccobiMT(int N, double delta, double* d_f, double* d_u, double* d_uold){
	 int j = blockIdx.y*blockDim.y+threadIdx.y;  //row thead id
     int i = blockIdx.x*blockDim.x+threadIdx.x;  //column thread id
     if (i==0 || j==0 || i==N-1 || j==N-1) return;
	 if ( i < N ){
	 	d_u[i*N +j] = 0.25 *(d_uold[(i-1)*N +j] + d_uold[(i+1)*N +j] + d_uold[i*N + j-1] + d_uold[i*N + j+1] + delta*delta*d_f[i*N +j]);
	 }
}

int poisson_gpu1(int N, int kmax, double delta, double* h_f, double* h_u){

	hipSetDevice(3);
	//allocate memory in device
	double* d_u;    hipMalloc((void**)&d_u, N*N*sizeof(double));
	double* d_uold; hipMalloc((void**)&d_uold, N*N*sizeof(double));
	double* d_f;    hipMalloc((void**)&d_f, N*N*sizeof(double));
	          

	// Transfer data from host to device
	hipMemcpy(d_u, h_u, N*N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_f, h_f, N*N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_uold, h_u, N*N*sizeof(double), hipMemcpyHostToDevice);

	int iterations = 0;
	// Kernel launch
	while( iterations < kmax) {

		jaccobi<<<1,1>>>(N,  delta,  d_f,  d_u, d_uold);
		checkCudaErrors(hipDeviceSynchronize());

		double* tmp;
		tmp = d_u;
		d_u = d_uold;
		d_uold = tmp;

		iterations++;
	}

	 // Transfer results from device to host
	 hipMemcpy(h_u, d_u, N*N*sizeof(double), hipMemcpyDeviceToHost);

	//print matrix d_C
	//print_matrix( m,  n, d_C);

	// Free memory
	hipFree(d_u);
	hipFree(d_uold);
 	hipFree(d_f);
 	return(kmax);
}

int poisson_gpu2(int N, int kmax, double delta, double* h_f, double* h_u){

	hipSetDevice(3);
	//allocate memory in device
	double* d_u;    hipMalloc((void**)&d_u, N*N*sizeof(double));
	double* d_uold; hipMalloc((void**)&d_uold, N*N*sizeof(double));
	double* d_f;    hipMalloc((void**)&d_f, N*N*sizeof(double));
	          

	// Transfer data from host to device
	hipMemcpy(d_u, h_u, N*N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_f, h_f, N*N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_uold, h_u, N*N*sizeof(double), hipMemcpyHostToDevice);

    dim3 NUM_BLOCKS = dim3( N/32 +1 , N/32 +1 );
    dim3 NUM_THREADS = dim3( 32, 32);

    if (N <= 32 )
    {
    	NUM_BLOCKS = dim3( 1, 1, 1 );
    	NUM_THREADS = dim3( N, N, 1);
    }

	int iterations = 0;

	// Kernel launch
	while( iterations < kmax) {

		jaccobiMT<<< NUM_BLOCKS, NUM_THREADS >>>(N,  delta,  d_f,  d_u, d_uold);
		checkCudaErrors(hipDeviceSynchronize());

		double* tmp;
		tmp = d_u;
		d_u = d_uold;
		d_uold = tmp;

		iterations++;
	}

	 // Transfer results from device to host
	 hipMemcpy(h_u, d_u, N*N*sizeof(double), hipMemcpyDeviceToHost);

	//print matrix d_C
	//print_matrix( m,  n, d_C);

	// Free memory
	hipFree(d_u);
	hipFree(d_uold);
 	hipFree(d_f);

 	return(kmax);
}
